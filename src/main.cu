#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <cassert>

#include "stb_image.h"
#include "stb_image_write.h"

using namespace std;

typedef struct Pixel{
    unsigned char r, g, b, a;
} Pixel;

void ConvertImageToGrayCPU(char* imageRGBA, int width, int height){
    for(int i=0;i<height;i++){
        for(int j=0;j<width;j++){
            Pixel* pixel = (Pixel*)&imageRGBA[(i*width)*4+4*j];
            float pixelValue = pixel->r*0.299f + pixel->g*0.587f + pixel->b*0.114f;
            unsigned char gray = (unsigned char)pixelValue;
            pixel->r = gray;
            pixel->g = gray;
            pixel->b = gray;
            pixel->a = 255;
        }
    }
} 

__global__ void ConvertImageToGrayGPU(unsigned char* imageRGBA, int width, int height) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uint32_t idx = (y * width + x) * 4;
        Pixel* pixel = (Pixel*)&imageRGBA[idx];
        float pixelValue = pixel->r * 0.299f + pixel->g * 0.587f + pixel->b * 0.114f;
        unsigned char gray = (unsigned char)pixelValue;
        pixel->r = gray;
        pixel->g = gray;
        pixel->b = gray;
        pixel->a = 255;
    }
}

int main(){
    // Hardcoded image filename
    const char* filename = "data/sample_data_color.jpg";

    // Open image
    int width, height, componentCount;
    unsigned char* imageData = stbi_load(filename, &width, &height, &componentCount, 0);
    if(!imageData){
        cout << "Failed to open \"" << filename << "\"\n";
        return -1;
    }

    // Validate image size
    if(width % 32 != 0 || height % 32 != 0){
        // Image size must be a multiple of 32
        cout << "Image size must be a multiple of 32\n";
        stbi_image_free(imageData);
        return -1;
    }

    /*
    cout << "Image size: " << width << "x" << height << "\n";
    cout << "Loading....." << "\n";
    cout << "Processing the image....." << "\n";
    ConvertImageToGrayCPU((char*)imageData, width, height);
    cout << "DONE" << "\n"; 
    */
    //copy data to device

    cout << "Copying Data to GPU" << "\n";
    unsigned char* ptrImageDataGpu = nullptr;
    assert(hipMalloc(&ptrImageDataGpu, width*height*4)== hipSuccess);
    assert(hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice)==hipSuccess);
    cout << "DONE" << "\n";


    cout << "Running CUDA Kernel.." << "\n";
    dim3 blockSize(32,32);
    dim3 gridSize(width/blockSize.x, height/blockSize.y);
    ConvertImageToGrayGPU<<<gridSize, blockSize>>>(ptrImageDataGpu, width, height);
    cout << "DONE" << "\n";

    //copying data from gpu to host
    cout << "Copying Data from GPY.." << "\n";
    assert(hipMemcpy(imageData,ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost)==hipSuccess);

    // Building output filename
    string outputFilename = filename;
    size_t dotPos = outputFilename.find_last_of('.');
    if (dotPos != string::npos) {
        outputFilename = "data/" + outputFilename.substr(outputFilename.find_last_of('/') + 1, dotPos) + "_gray.jpg";
    } else {
        outputFilename = "data/" + outputFilename.substr(outputFilename.find_last_of('/') + 1) + "_gray.jpg";
    }

    // Write image back
    stbi_write_jpg(outputFilename.c_str(), width, height, 4, imageData, 100);

    // Close 
    stbi_image_free(imageData);

    cout << "Converted image saved as \"" << outputFilename << "\"\n";
    cout << "DONE" << "\n";

    return 0;
}
